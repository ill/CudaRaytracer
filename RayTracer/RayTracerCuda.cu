#include "hip/hip_runtime.h"
#include "illEngine/Graphics/serial/Camera/Camera.h"

#include "outputTga.h"
#include "RayTracerCuda.h"
#include "util.h"
#include "cudaKernels.h"



RayTracerCuda::RayTracerCuda(const glm::uvec2& resolution) : RayTracerBase(resolution) {

    m_colorBuffer = new uint32_t[m_resolution.x * m_resolution.y];

    //create all the spheres
    for(unsigned int x = 0; x < 10; x++) {
        for(unsigned int y = 0; y < 10; y++) {
            for(unsigned int z = 0; z < 10; z++) {
                m_spheres.push_back(SphereData());

                m_spheres.back().m_color = glm::vec4(0.1f + (float) x / 10.0f, 0.1f + (float) y / 10.0f, 0.1f + (float) z / 10.0f, 1.0f);
                m_spheres.back().m_sphere.m_radius = 3.0f;
                m_spheres.back().m_sphere.m_center = glm::vec3(10.0f * x, 10.0f * y, 10.0f *z);
            }
        }
    }

    //create the lights

    //a light white light
    m_lights.push_back(SphereData());

    m_lights.back().m_color = glm::vec4(1.0f, 1.0f, 1.0f, 1.0f);
    m_lights.back().m_sphere.m_radius = 100.0f;
    m_lights.back().m_sphere.m_center = glm::vec3(-30.0f, 50.0f, 50.0f);

    //a red light
    m_lights.push_back(SphereData());

    m_lights.back().m_color = glm::vec4(1.0f, 0.0f, 0.0f, 1.0f);
    m_lights.back().m_sphere.m_radius = 30.0f;
    m_lights.back().m_sphere.m_center = glm::vec3(50.0f, 50.0f, -10.0f);

    //a blue light
    m_lights.push_back(SphereData());

    m_lights.back().m_color = glm::vec4(0.0f, 0.0f, 1.0f, 1.0f);
    m_lights.back().m_sphere.m_radius = 30.0f;
    m_lights.back().m_sphere.m_center = glm::vec3(50.0f, -10.0f, 50.0f);
}

RayTracerCuda::~RayTracerCuda() {
   delete[] m_colorBuffer;
}


/*__device__ RayTracerCuda::SphereData * RayTracerCuda::sphereForRay(const glm::vec3& rayOrigin, const glm::vec3& rayDirection, glm::mediump_float& distance, const SphereData* omitSphere) {
    //find the closest sphere
    const SphereData * closestSphere = NULL;

    for (std::vector<SphereData>::const_iterator iter = m_spheres.begin(); iter != m_spheres.end(); iter++) {
        const SphereData& sphere = *iter;

        glm::mediump_float thisDistance;

        //do a ray sphere intersection
        if(sphere.m_sphere.rayIntersection(rayOrigin, rayDirection, thisDistance)) {
            //if this is the closest sphere so far, choose this bill
            if(&sphere != omitSphere &&(closestSphere == NULL || thisDistance < distance)) {
                distance = thisDistance;
                closestSphere = &sphere;
            }
        }
    }

    return closestSphere;
}*/

void RayTracerCuda::rayTraceScene(const illGraphics::Camera& camera) const {
   uint32_t* colorBufferD;
   RayTracerBase::SphereData* spheresD;
   RayTracerBase::SphereData* lightsD;
   const RayTracerBase::SphereData* tmpSpheres = &m_spheres[0];
   const RayTracerBase::SphereData* tmpLights = &m_lights[0];
   glm::vec3 a;
   glm::vec3 b;
   Scene scene;

   // Allocate device memory
   hipMalloc((void **)&colorBufferD, m_resolution.x * m_resolution.y * sizeof(uint32_t));
   hipMalloc((void **)&spheresD, m_spheres.size() * sizeof(SphereData));
   hipMalloc((void **)&lightsD, m_lights.size() * sizeof(SphereData));
   hipMemcpy(spheresD, tmpSpheres, m_spheres.size() * sizeof(SphereData), hipMemcpyHostToDevice);
   hipMemcpy(lightsD, tmpLights, m_lights.size() * sizeof(SphereData), hipMemcpyHostToDevice);

   // Initialize things for kernel
   Camera_t camera_t;
   camera_t.m_transform = camera.getTransform();
   //camera_t.m_frustum = camera.getViewFrustum();
   camera_t.m_modelView = camera.getModelView();
   camera_t.m_projection = camera.getProjection();
   camera_t.m_canonical = camera.getCanonical();
   scene.camera = camera_t;
   scene.colorBuffer = colorBufferD;
   scene.xRes = m_resolution.x;
   scene.yRes = m_resolution.y;
   scene.spheres = spheresD;
   scene.numSpheres = m_spheres.size();
   scene.lights = lightsD;
   scene.numLights = m_lights.size();

   // Set up grid and block dimensions
   dim3 dimGrid(ceil(m_resolution.x / 32), ceil(m_resolution.y / 32));
   dim3 dimBlock(BLOCK_WIDTH, BLOCK_HEIGHT);

   // Call kernel
   //RTkernel<<<dimGrid, dimBlock>>>(camera_t, colorBufferD, m_resolution.x, m_resolution.y, spheresD, lightsD);
   RTkernel<<<dimGrid, dimBlock>>>(scene);

   // Retrieve results
   hipMemcpy(m_colorBuffer, colorBufferD, m_resolution.x * m_resolution.y * sizeof(uint32_t), hipMemcpyDeviceToHost);

   // Clean up, free data from global memory
}

void RayTracerCuda::output(const char * fileName) const {
    tgaOut(m_colorBuffer, m_resolution, fileName);
}
